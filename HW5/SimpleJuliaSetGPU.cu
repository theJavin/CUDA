#include "hip/hip_runtime.h"
//nvcc SimpleJuliaSetGPU.cu -o SimpleJuliaSetGPU -lglut -lGL -lm
// This is a simple Julia set which is repeated iterations of 
// Znew = Zold + C whre Z and Care imaginary numbers.
// After so many tries if Zinitial escapes color it black if it stays around color it red.

#include <GL/glut.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "../hipError_t.h"

void Init();


#define A  -0.824  //real
#define B  -0.1711   //imaginary

unsigned int window_width = 1024;
unsigned int window_height = 1024;

int N = window_height*window_width;

dim3 BlockSize; //This variable will hold the Dimensions of your block
dim3 GridSize; //This variable will hold the Dimensions of your grid

float *pixels, *pixelsGPU; 

void SetUpCudaDevices()
{
	BlockSize.x = 1024;
	BlockSize.y = 1;
	BlockSize.z = 1;
	
	GridSize.x = 1024;
	GridSize.y = 1;
	GridSize.z = 1;
}

float xMin = -2.0;
float xMax =  2.0;
float yMin = -2.0;
float yMax =  2.0;

float stepSizeX = (xMax - xMin)/((float)window_width);
float stepSizeY = (yMax - yMin)/((float)window_height);

__global__ void compute(float *pixels, float stepSizeX, float stepSizeY, float xMin, float yMin, int N) 
{
	int id = 3*(threadIdx.x + blockDim.x*blockIdx.x); 
	float mag = 0.0;
	float maxmag = 10.0;
	float temp;
	int count = 0;
	int maxCount = 1000;

	float x = xMin + stepSizeX*threadIdx.x;
	float y = yMin + stepSizeY*blockIdx.x;

	if(id<3*N)
	{
		while(count < maxCount)
		{
			temp = x;
			x = x*x-y*y + A;
			y = (2.0*temp*y) + B;
			
			mag += sqrt(x*x + y*y);
			count++;
		}
		if(mag > maxmag)
		{
			pixels[id] = 1.0;
			pixels[id+1] = 0.0;
			pixels[id+2] = 0.0;
		}
		else
		{
			pixels[id] = 0.0;
			pixels[id+1] = 0.0;
			pixels[id+2] = 0.0;
		}
	}
}

void Display()
{
	Init();
	errorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(pixelsGPU, pixels, N*3*sizeof(float), hipMemcpyHostToDevice);
	errorCheck(__FILE__, __LINE__);
	compute<<<GridSize, BlockSize>>>(pixelsGPU, stepSizeX, stepSizeY, xMin, yMin, N);
	errorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(pixels, pixelsGPU, N*3*sizeof(float), hipMemcpyDeviceToHost);
	errorCheck(__FILE__, __LINE__);
	glDrawPixels(window_width, window_height, GL_RGB, GL_FLOAT, pixels); 
	glFlush(); 
}

void Init()
{
	pixels = (float*)malloc(N*3*sizeof(float));
	hipMalloc(&pixelsGPU, N*3*sizeof(float));
}

void CleanUp()
{
	free(pixels);
	hipFree(pixelsGPU);
}

int main(int argc, char** argv)
{ 
	SetUpCudaDevices();
	
   	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
   	glutInitWindowSize(window_width, window_height);
	glutCreateWindow("Fractals man, fractals.");
//	Init();
   	glutDisplayFunc(Display);	
	CleanUp();
   	glutMainLoop();
}
